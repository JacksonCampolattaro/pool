#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <torch/library.h>

using torch::Tensor;


constexpr uint64_t group_size = 8;
constexpr uint64_t block = 512;

template <typename T>
struct __builtin_align__(group_size * sizeof(T)) value_group {
  T v[group_size];
};

// Generic comparison function
template <typename T>
__device__ inline bool is_greater(const T &a, const T &b) {
  return a > b;
}

// Specialization for half precision
template <>
__device__ inline bool is_greater<half>(const half &a, const half &b) {
  return __hgt(a, b);
}

// Generic zero value
template <typename T> __device__ inline T zero_value() { return T(0); }

// Specialization for half precision
template <> __device__ inline half zero_value<half>() {
  return __int2half_rz(0);
}

template <typename T>
__global__ void __launch_bounds__(block)
    maxpool_forward_kernel(T *output, uint32_t *indices, const T *feature,
                           const uint64_t *knn, const uint64_t k,
                           const uint64_t N, const uint64_t C_,
                           const uint64_t NC) {
  // idx = bNC + nC + c
  const uint64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= NC)
    return;
  const uint64_t C = C_ / group_size;
  // bN + n
  const uint64_t BN = idx / C;
  const uint64_t n = BN % N;
  // feature base idx : bNC_ + c*group_size, striding C_
  const uint64_t feature_base = (BN - n) * C_ + (idx % C) * group_size;
  // knn base idx : bNk + nk, striding 1
  uint64_t knn_idx = BN * k;
  const uint64_t knn_end = knn_idx + k;
  uint64_t nbr_idx = knn[knn_idx];

  value_group<T> max_val =
      *(value_group<T> *)(feature + feature_base + nbr_idx * C_);
  uint32_t max_idx[group_size];
  for (uint64_t f_idx = 0; f_idx < group_size; ++f_idx) {
    max_idx[f_idx] = nbr_idx;
  }

  for (++knn_idx; knn_idx < knn_end; ++knn_idx) {
    nbr_idx = knn[knn_idx];
    const value_group<T> valn =
        *(value_group<T> *)(feature + feature_base + nbr_idx * C_);
    for (uint64_t f_idx = 0; f_idx < group_size; ++f_idx) {
      const T val = valn.v[f_idx];
      if (is_greater(val, max_val.v[f_idx])) {
        max_val.v[f_idx] = val;
        max_idx[f_idx] = nbr_idx;
      }
    }
  }

  const value_group<T> valn =
      *(value_group<T> *)(feature + feature_base + n * C_);
  for (uint64_t f_idx = 0; f_idx < group_size; ++f_idx) {
    indices[feature_base + n * C_ + f_idx] = max_idx[f_idx];
  }

  *(value_group<T> *)(output + feature_base + n * C_) = max_val;
  //    const uint64_t output_base = feature_base + n * C_;
  //    if constexpr (std::is_same_v<T, at::Half>) {
  //        // Special handling for half precision
  //        for (uint64_t f_idx = 0; f_idx < group_size/2; ++f_idx)
  //            *reinterpret_cast<half2*>(output + output_base + f_idx*2) =
  //                __halves2half2(max_val.v[f_idx*2], max_val.v[f_idx*2+1]);
  //
  //    } else {
  //        // Default handling for other types
  //        *(value_group<T>*)(output + output_base) = max_val;
  //    }
}

void maxpool_forward_inplace(Tensor &output, Tensor &indices,
                             const Tensor &feature, const Tensor &knn) {
  const uint64_t N = knn.size(0);
  const uint64_t k = knn.size(1);
  const uint64_t C = output.size(1);
  const uint64_t NC = 1 * N * (C / group_size);
  const uint64_t grid = (NC + block - 1) / block;

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::kHalf, at::kBFloat16, feature.scalar_type(), "maxpool_forward", [&] {
        maxpool_forward_kernel<scalar_t><<<grid, block>>>(
            output.data_ptr<scalar_t>(), indices.data_ptr<uint32_t>(),
            feature.data_ptr<scalar_t>(), (const uint64_t *)knn.data_ptr(), k,
            N, C, NC);
      });
}

template <typename T>
__global__ void __launch_bounds__(block)
    maxpool_infer_kernel(T *output, const T *feature, const uint64_t *knn,
                         const uint64_t k, const uint64_t N, const uint64_t C_,
                         const uint64_t NC) {
  // idx = nC + c
  const uint64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= NC)
    return;
  const uint64_t C = C_ / group_size;
  // bN + n
  const uint64_t BN = idx / C;
  const uint64_t n = BN % N;
  // feature base idx : bNC_ + c*group_size, striding C_
  const uint64_t feature_base = (BN - n) * C_ + (idx % C) * group_size;
  // knn base idx : bNk + nk, striding 1
  uint64_t knn_idx = BN * k;
  const uint64_t knn_end = knn_idx + k;
  uint64_t nbr_idx = knn[knn_idx];

  value_group<T> max_val =
      *(value_group<T> *)(feature + feature_base + nbr_idx * C_);

  for (++knn_idx; knn_idx < knn_end; ++knn_idx) {
    nbr_idx = knn[knn_idx];
    const value_group<T> valn =
        *(value_group<T> *)(feature + feature_base + nbr_idx * C_);
    for (uint64_t f_idx = 0; f_idx < group_size; ++f_idx) {
      const T val = valn.v[f_idx];
      if (is_greater(val, max_val.v[f_idx])) {
        max_val.v[f_idx] = val;
      }
    }
  }

  *(value_group<T> *)(output + feature_base + n * C_) = max_val;

  //    const uint64_t output_base = feature_base + n * C_;
  //    if constexpr (std::is_same_v<T, at::Half>) {
  //        for (uint64_t f_idx = 0; f_idx < group_size/2; ++f_idx)
  //            *reinterpret_cast<half2*>(output + output_base + f_idx*2) =
  //                __halves2half2(max_val.v[f_idx*2], max_val.v[f_idx*2+1]);
  //
  //    } else {
  //        *(value_group<T>*)(output + output_base) = max_val;
  //    }
}

void maxpool_infer_inplace(Tensor &output, const Tensor &feature,
                           const Tensor &knn) {
  const uint64_t k = knn.size(1);
  const uint64_t N = knn.size(0);
  const uint64_t C = output.size(1);
  const uint64_t NC = 1 * N * (C / group_size);
  const uint64_t grid = (NC + block - 1) / block;

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::kHalf, at::kBFloat16, feature.scalar_type(), "maxpool_infer", [&] {
        maxpool_infer_kernel<scalar_t><<<grid, block>>>(
            output.data_ptr<scalar_t>(), feature.data_ptr<scalar_t>(),
            (const uint64_t *)knn.data_ptr(), k, N, C, NC);
      });
}

template <typename T>
__global__ void maxpool_backward_kernel(T *output, const uint32_t *indices,
                                        const T *grad, const uint64_t N,
                                        const uint64_t C, const uint64_t NC) {
  const uint64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= NC)
    return;
  const uint64_t n = idx / C % N;
  const uint64_t backidx = indices[idx];
  const T g = grad[idx];
  const uint64_t feature_base = idx - n * C + backidx * C;

  if constexpr (std::is_same_v<T, at::Half>) {
    const uint64_t high = idx % 2;
    const half2 x = __halves2half2(
        high ? __int2half_rz(0) : *reinterpret_cast<const half *>(&g),
        high ? *reinterpret_cast<const half *>(&g) : __int2half_rz(0));
    atomicAdd(reinterpret_cast<half2 *>(output + feature_base - high), x);
  } else {
    atomicAdd(output + feature_base, g);
  }
}

void maxpool_backward_inplace(Tensor &output, const Tensor &indices,
                              const Tensor &grad) {
  const uint64_t M = indices.size(0);
  const uint64_t C = output.size(1);
  const uint64_t MC = M * C;
  const uint64_t grid = (MC + block - 1) / block;

  AT_DISPATCH_FLOATING_TYPES_AND(
      at::kHalf, output.scalar_type(), "maxpool_backward", [&] {
        maxpool_backward_kernel<scalar_t><<<grid, block>>>(
            output.data_ptr<scalar_t>(), indices.data_ptr<uint32_t>(),
            grad.data_ptr<scalar_t>(), M, C, MC);
      });
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("maxpool_forward_inplace", &maxpool_forward_inplace);
  m.def("maxpool_infer_inplace", &maxpool_infer_inplace);
  m.def("maxpool_backward_inplace", &maxpool_backward_inplace);
}


